#include "hip/hip_runtime.h"
/*!
 * \file gpu_reductions.h
 * \author Robert 'Bob' Caddy (rvc@pitt.edu)
 * \brief
 * \version 0.1
 * \date 2022-03-04
 *
 * \copyright Copyright (c) 2022
 *
 */

// Globals

#ifdef  CUDA_BUILD
    #define WARPSIZE 32
#endif  //CUDA_BUILD
#ifdef  HIP_BUILD
    #define WARPSIZE 64
#endif  //HIP_BUILD

static constexpr int maxWarpsPerBlock = 1024/WARPSIZE; // outside kernel
typedef double Real;

 // External Includes

// STL Includes
#include <stdio.h>
#include <cstdint>
#include <iostream>
#include <vector>
#include <string>

// Local Includes
// #include "sumReduction.h"
#include "maxReduction.h"
#include "timeStepTestOriginal.h"
#include "timeStepTestNew.h"

__global__ void checkDims()
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0)
    {
        int const tid  = threadIdx.x;
        int const bid  = blockIdx.x;
        int const bdim = blockDim.x;
        int const gdim = gridDim.x;

        printf("\nthreadIdx.x = %i", tid);
        printf("\nblockIdx.x  = %i", bid);
        printf("\nblockDim.x  = %i", bdim);
        printf("\ngridDim.x   = %i", gdim);
    }
}

int	main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    std::cout << "prop.maxThreadsPerMultiProcessor = " << prop.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "prop.multiProcessorCount         = " << prop.multiProcessorCount << std::endl;
    std::cout << "prop.maxThreadsPerBlock          = " << prop.maxThreadsPerBlock << std::endl;

    int numThreads = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount;
    int numBlocks  = numThreads / prop.maxThreadsPerBlock;

    std::cout << std::endl;
    std::cout << "numThreads = " << numThreads << std::endl;
    std::cout << "numBlocks  = " << numBlocks << std::endl;

    // Testing variables
    int const trials   = 1000;
    int const gridSize = 512;

    std::cout << std::endl;
    std::cout << "number of trials = " << trials   << std::endl;
    std::cout << "Grid size        = " << gridSize << std::endl;

    // output variables
    Real maxReducedAtomic, maxReduced, oldDTI, newDTI;

    std::cout << std::endl;
    maxReducedAtomic = gpuAtomicMaxReduction(trials, gridSize);

    std::cout << std::endl;
    maxReduced = gpuMaxReduction(trials, gridSize);

    std::cout << std::endl;
    oldDTI = calcDtiOriginal(trials, gridSize);

    std::cout << std::endl;
    newDTI = calcDtiNEW(trials, gridSize);

    std::cout << std::endl;
    std::cout << "maxReducedAtomic = " << maxReducedAtomic << std::endl;
    std::cout << "maxReduced       = " << maxReduced       << std::endl;
    std::cout << "oldDTI           = " << oldDTI           << std::endl;
    std::cout << "newDTI           = " << newDTI           << std::endl;

    return 0;
}
